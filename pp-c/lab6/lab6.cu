/*Author: Hunter Esler
 * Course: CSCI 4330
 * Lab number: Lab 6
 * Purpose: This lab will use mpich to calculage integral of sqrt(4-x^2) from 0 to 2 using trapezoidal rule with cuda
 * Due date: 4/22/2019
 * */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

//device function
__device__
double func(double x) {
	return (sqrtf(4-x*x));
}

//kernel, grabs left and right of step and uses the custom device func and then stores area in its spot
__global__
void calcarea(float* step, float* areas) {
	
	int here = threadIdx.x;

	float left, right;
	left = here * (*step);
	right = (*step) * (here + 1);
	
	//do ttrapezoid rule
	areas[here] = ((func(left)+func(right))*(*step))/2.0;
}


int main(int argc, char * argv[]) {
	int nproc, i;
	float* step = (float*)malloc(sizeof(float));//the step for each cuda core
	float* stepgpu;//gpu pointer
	float* areasgpu;//gpu pointer
	float* areas;//return areas from gpu
	float sum = 0.0;//total sum

	//grabbing number of steps
	printf("Enter steps: ");
	scanf("%d", &nproc);

	areas = (float*)malloc(nproc*sizeof(float));
	*step = 2.0 / nproc;
	printf("Mallocing and starting function on gpu\n");
	//mallocing on gpu
	hipMalloc((void**)&stepgpu, sizeof(float));
	hipMalloc((void**)&areasgpu, nproc * sizeof(float));
	//giving gpu step size
	hipMemcpy(stepgpu, step, sizeof(float), hipMemcpyHostToDevice);

	//calling kernel
	//printf("Starting function on gpu\n");
	calcarea<<<1, nproc>>>(stepgpu, areasgpu);

	//copying from gpu to cpu
	hipMemcpy(areas, areasgpu, nproc * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(stepgpu);
	hipFree(areasgpu);
	free(step);
	
	//sum and print result
	for (i = 0; i < nproc; i++) {
		sum+=areas[i];
	}
	printf("sum: %f\n", sum);

	free(areas);
	return 0;
}
